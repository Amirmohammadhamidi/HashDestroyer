#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "MD5GPUKernel.h"
#include "MD5Device.h"  // Include our __device__ MD5 function

// Kernel: generate candidate from idx (as a base-36 number) and compute its MD5.
__global__ void md5BruteForceKernel(const char* d_targetHash, char* d_foundCandidate, bool* d_found, int numCandidates) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numCandidates || *d_found)
        return;
    
    // Convert idx to a 5-character candidate using Base36.
    char candidate[6];
    const char charset[37] = "abcdefghijklmnopqrstuvwxyz0123456789";
    int base = 36;
    int temp = idx;
    candidate[5] = '\0';
    for (int i = 4; i >= 0; i--) {
        candidate[i] = charset[temp % base];
        temp /= base;
    }
    
    // Compute MD5 on candidate.
    char computedHash[33];
    deviceMD5(candidate, computedHash);
    
    // Compare computed hash with target hash.
    bool match = true;
    for (int i = 0; i < 32; i++) {
        if (computedHash[i] != d_targetHash[i]) {
            match = false;
            break;
        }
    }
    
    if (match) {
        // Copy candidate to global memory.
        for (int i = 0; i < 5; i++)
            d_foundCandidate[i] = candidate[i];
        d_foundCandidate[5] = '\0';
        *d_found = true;
    }
}

extern "C" void runMD5BruteForceKernel(const char* targetHash, char* foundCandidate, bool* found, int numCandidates) {
    size_t hashSize = 33 * sizeof(char);
    char* d_targetHash;
    char* d_foundCandidate;
    bool* d_found;
    hipMalloc(&d_targetHash, hashSize);
    hipMalloc(&d_foundCandidate, 6 * sizeof(char));
    hipMalloc(&d_found, sizeof(bool));
    
    hipMemcpy(d_targetHash, targetHash, hashSize, hipMemcpyHostToDevice);
    bool foundInit = false;
    hipMemcpy(d_found, &foundInit, sizeof(bool), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int gridSize = (numCandidates + blockSize - 1) / blockSize;
    md5BruteForceKernel<<<gridSize, blockSize>>>(d_targetHash, d_foundCandidate, d_found, numCandidates);
    hipDeviceSynchronize();
    
    bool h_found;
    hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
    if (h_found) {
        hipMemcpy(foundCandidate, d_foundCandidate, 6 * sizeof(char), hipMemcpyDeviceToHost);
        *found = true;
    }
    
    hipFree(d_targetHash);
    hipFree(d_foundCandidate);
    hipFree(d_found);
}